#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#include <algorithm>
#include "convert_csp.h"
#include "NVEncFilterSubburn.h"
#include "NVEncParam.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#pragma warning (pop)
#include "rgy_cuda_util.h"

#if ENABLE_AVSW_READER && ENABLE_LIBASS_SUBBURN

static __device__ float lerpf(float a, float b, float c) {
    return a + (b - a) * c;
}

template<typename TypePixel, int bit_depth>
__inline__ __device__
TypePixel blend(TypePixel pix, uint8_t alpha, uint8_t val, float transparency_offset, float pix_offset, float contrast) {
    //alpha値は 0が透明, 255が不透明
    float subval = val * (1.0f / (float)(1 << 8));
    subval = contrast * (subval - 0.5f) + 0.5f + pix_offset;
    float ret = lerpf((float)pix, subval * (float)(1<<bit_depth), alpha * (1.0f / 255.0f) * (1.0f - transparency_offset));
    return (TypePixel)clamp(ret, 0.0f, (1<<bit_depth)-0.5f);
}

template<typename TypePixel2, int bit_depth>
__inline__ __device__
void blend(void *pix, const void *alpha, const void *val, float transparency_offset, float pix_offset, float contrast) {
    uchar2 a = *(uchar2 *)alpha;
    uchar2 v = *(uchar2 *)val;
    TypePixel2 p = *(TypePixel2 *)pix;
    p.x = blend<decltype(TypePixel2::x), bit_depth>(p.x, a.x, v.x, transparency_offset, pix_offset, contrast);
    p.y = blend<decltype(TypePixel2::x), bit_depth>(p.y, a.y, v.y, transparency_offset, pix_offset, contrast);
    *(TypePixel2 *)pix = p;
}

template<typename TypePixel, int bit_depth, bool yuv420>
__global__ void kernel_subburn(
    uint8_t *__restrict__ pPlaneY,
    uint8_t *__restrict__ pPlaneU,
    uint8_t *__restrict__ pPlaneV,
    const int pitchFrameY,
    const int pitchFrameU,
    const int pitchFrameV,
    const uint8_t *__restrict__ pSubY, const uint8_t *__restrict__ pSubU, const uint8_t *__restrict__ pSubV, const uint8_t *__restrict__ pSubA,
    const int pitchSub,
    const int width, const int height, bool interlaced, float transparency_offset, float brightness, float contrast) {
    //縦横2x2pixelを1スレッドで処理する
    const int ix = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    const int iy = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

    struct __align__(sizeof(TypePixel) * 2) TypePixel2 {
        TypePixel x, y;
    };
    if (ix < width && iy < height) {
        pPlaneY += iy * pitchFrameY + ix * sizeof(TypePixel);
        pSubY   += iy * pitchSub + ix;
        pSubU   += iy * pitchSub + ix;
        pSubV   += iy * pitchSub + ix;
        pSubA   += iy * pitchSub + ix;

        blend<TypePixel2, bit_depth>(pPlaneY,               pSubA,            pSubY,            transparency_offset, brightness, contrast);
        blend<TypePixel2, bit_depth>(pPlaneY + pitchFrameY, pSubA + pitchSub, pSubY + pitchSub, transparency_offset, brightness, contrast);

        if (yuv420) {
            pPlaneU += (iy>>1) * pitchFrameU + (ix>>1) * sizeof(TypePixel);
            pPlaneV += (iy>>1) * pitchFrameV + (ix>>1) * sizeof(TypePixel);
            uint8_t subU, subV, subA;
            if (interlaced) {
                if (((iy>>1) & 1) == 0) {
                    const int offset_y1 = (iy+2<height) ? pitchSub*2 : 0;
                    subU = (pSubU[0] * 3 + pSubU[offset_y1] + 2) >> 2;
                    subV = (pSubV[0] * 3 + pSubV[offset_y1] + 2) >> 2;
                    subA = (pSubA[0] * 3 + pSubA[offset_y1] + 2) >> 2;
                } else {
                    subU = (pSubU[-pitchSub] + pSubU[pitchSub] * 3 + 2) >> 2;
                    subV = (pSubV[-pitchSub] + pSubV[pitchSub] * 3 + 2) >> 2;
                    subA = (pSubA[-pitchSub] + pSubA[pitchSub] * 3 + 2) >> 2;
                }
            } else {
                subU = (pSubU[0] + pSubU[pitchSub] + 1) >> 1;
                subV = (pSubV[0] + pSubV[pitchSub] + 1) >> 1;
                subA = (pSubA[0] + pSubA[pitchSub] + 1) >> 1;
            }
            *(TypePixel *)pPlaneU = blend<TypePixel, bit_depth>(*(TypePixel *)pPlaneU, subA, subU, transparency_offset, 0.0f, 1.0f);
            *(TypePixel *)pPlaneV = blend<TypePixel, bit_depth>(*(TypePixel *)pPlaneV, subA, subV, transparency_offset, 0.0f, 1.0f);
        } else {
            pPlaneU += iy * pitchFrameU + ix * sizeof(TypePixel);
            pPlaneV += iy * pitchFrameV + ix * sizeof(TypePixel);
            blend<TypePixel2, bit_depth>(pPlaneU,               pSubA,            pSubU,            transparency_offset, 0.0f, 1.0f);
            blend<TypePixel2, bit_depth>(pPlaneU + pitchFrameU, pSubA + pitchSub, pSubU + pitchSub, transparency_offset, 0.0f, 1.0f);
            blend<TypePixel2, bit_depth>(pPlaneV,               pSubA,            pSubV,            transparency_offset, 0.0f, 1.0f);
            blend<TypePixel2, bit_depth>(pPlaneV + pitchFrameV, pSubA + pitchSub, pSubV + pitchSub, transparency_offset, 0.0f, 1.0f);
        }
    }
}

template<typename TypePixel, int bit_depth>
RGY_ERR proc_frame(RGYFrameInfo *pFrame,
    const RGYFrameInfo *pSubImg,
    int pos_x, int pos_y,
    float transparency_offset, float brightness, float contrast,
    hipStream_t stream) {
    //焼きこみフレームの範囲内に収まるようチェック
    const int burnWidth  = std::min((pos_x & ~1) + pSubImg->width,  pFrame->width)  - (pos_x & ~1);
    const int burnHeight = std::min((pos_y & ~1) + pSubImg->height, pFrame->height) - (pos_y & ~1);
    if (burnWidth <= 0 || burnHeight <= 0) {
        return RGY_ERR_NONE;
    }

    dim3 blockSize(32, 8);
    dim3 gridSize(divCeil(burnWidth, blockSize.x * 2), divCeil(burnHeight, blockSize.y * 2)); // 2x2pixel/thread
    auto planeFrameY = getPlane(pFrame, RGY_PLANE_Y);
    auto planeFrameU = getPlane(pFrame, RGY_PLANE_U);
    auto planeFrameV = getPlane(pFrame, RGY_PLANE_V);
    auto planeSubY = getPlane(pSubImg, RGY_PLANE_Y);
    auto planeSubU = getPlane(pSubImg, RGY_PLANE_U);
    auto planeSubV = getPlane(pSubImg, RGY_PLANE_V);
    auto planeSubA = getPlane(pSubImg, RGY_PLANE_A);

    const int subPosX_Y = (pos_x & ~1);
    const int subPosY_Y = (pos_y & ~1);
    const int subPosX_UV = (RGY_CSP_CHROMA_FORMAT[pFrame->csp] == RGY_CHROMAFMT_YUV420) ? (pos_x >> 1) : (pos_x & ~1);
    const int subPosY_UV = (RGY_CSP_CHROMA_FORMAT[pFrame->csp] == RGY_CHROMAFMT_YUV420) ? (pos_y >> 1) : (pos_y & ~1);
    const int frameOffsetByteY = subPosY_Y  * planeFrameY.pitch[0] + subPosX_Y  * sizeof(TypePixel);
    const int frameOffsetByteU = subPosY_UV * planeFrameU.pitch[0] + subPosX_UV * sizeof(TypePixel);
    const int frameOffsetByteV = subPosY_UV * planeFrameV.pitch[0] + subPosX_UV * sizeof(TypePixel);

    if (   planeSubY.pitch[0] != planeSubU.pitch[0]
        || planeSubY.pitch[0] != planeSubV.pitch[0]
        || planeSubY.pitch[0] != planeSubA.pitch[0]) {
        return RGY_ERR_UNSUPPORTED;
    }

    hipError_t cudaerr = hipSuccess;
    if (RGY_CSP_CHROMA_FORMAT[pFrame->csp] == RGY_CHROMAFMT_YUV420) {
        kernel_subburn<TypePixel, bit_depth, true> << <gridSize, blockSize, 0, stream >> > (
            planeFrameY.ptr[0] + frameOffsetByteY,
            planeFrameU.ptr[0] + frameOffsetByteU,
            planeFrameV.ptr[0] + frameOffsetByteV,
            planeFrameY.pitch[0],
            planeFrameU.pitch[0],
            planeFrameV.pitch[0],
            planeSubY.ptr[0], planeSubU.ptr[0], planeSubV.ptr[0], planeSubA.ptr[0], planeSubY.pitch[0],
            burnWidth, burnHeight, interlaced(*pFrame), transparency_offset, brightness, contrast);
    } else {
        kernel_subburn<TypePixel, bit_depth, false> << <gridSize, blockSize, 0, stream >> > (
            planeFrameY.ptr[0] + frameOffsetByteY,
            planeFrameU.ptr[0] + frameOffsetByteU,
            planeFrameV.ptr[0] + frameOffsetByteV,
            planeFrameY.pitch[0],
            planeFrameU.pitch[0],
            planeFrameV.pitch[0],
            planeSubY.ptr[0], planeSubU.ptr[0], planeSubV.ptr[0], planeSubA.ptr[0], planeSubY.pitch[0],
            burnWidth, burnHeight, interlaced(*pFrame), transparency_offset, brightness, contrast);
    }
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return err_to_rgy(cudaerr);
    }
    return RGY_ERR_NONE;
}

SubImageData NVEncFilterSubburn::textRectToImage(const ASS_Image *image, hipStream_t stream) {
    //YUV420の関係で縦横2pixelずつ処理するので、2で割り切れている必要がある
    const int x_offset = ((image->dst_x % 2) != 0) ? 1 : 0;
    const int y_offset = ((image->dst_y % 2) != 0) ? 1 : 0;
    RGYFrameInfo img;
    img.csp = RGY_CSP_YUVA444;
    img.width  = ALIGN(image->w + x_offset, 2);
    img.height = ALIGN(image->h + y_offset, 2);
    img.mem_type = RGY_MEM_TYPE_CPU;
    img.picstruct = RGY_PICSTRUCT_FRAME;
    auto bufCPU = std::make_unique<CUFrameBuf>(img);
    auto err = bufCPU->allocHost();
    if (err != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("Failed to allocate host memory for subtitle image %dx%d: %s.\n"), image->w, image->h, get_err_mes(err));
        return SubImageData(
            std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
            std::unique_ptr<CUFrameBuf>(), 0, 0);
    }

    auto planeY = getPlane(&bufCPU->frame, RGY_PLANE_Y);
    auto planeU = getPlane(&bufCPU->frame, RGY_PLANE_U);
    auto planeV = getPlane(&bufCPU->frame, RGY_PLANE_V);
    auto planeA = getPlane(&bufCPU->frame, RGY_PLANE_A);

    //とりあえずすべて0で初期化しておく
    memset(planeY.ptr[0], 0, (size_t)planeY.pitch[0] * planeY.height);

    //とりあえずすべて0で初期化しておく
    //Alpha=0で透明なので都合がよい
    memset(planeA.ptr[0], 0, (size_t)planeA.pitch[0] * planeA.height);

    for (int j = 0; j < planeU.height; j++) {
        auto ptr = planeU.ptr[0] + j * planeU.pitch[0];
        for (int i = 0; i < planeU.pitch[0] / (int)sizeof(ptr[0]); i++) {
            ptr[i] = 128;
        }
    }
    for (int j = 0; j < planeV.height; j++) {
        auto ptr = planeV.ptr[0] + j * planeV.pitch[0];
        for (int i = 0; i < planeV.pitch[0] / (int)sizeof(ptr[0]); i++) {
            ptr[i] = 128;
        }
    }

    const uint32_t subColor = image->color;
    const uint8_t subR = (uint8_t) (subColor >> 24);
    const uint8_t subG = (uint8_t)((subColor >> 16) & 0xff);
    const uint8_t subB = (uint8_t)((subColor >>  8) & 0xff);
    const uint8_t subA = (uint8_t)(255 - (subColor        & 0xff));

    const uint8_t subY = (uint8_t)clamp((( 66 * subR + 129 * subG +  25 * subB + 128) >> 8) +  16, 0, 255);
    const uint8_t subU = (uint8_t)clamp(((-38 * subR -  74 * subG + 112 * subB + 128) >> 8) + 128, 0, 255);
    const uint8_t subV = (uint8_t)clamp(((112 * subR -  94 * subG -  18 * subB + 128) >> 8) + 128, 0, 255);

    //YUVで字幕の画像データを構築
    for (int j = 0; j < image->h; j++) {
        for (int i = 0; i < image->w; i++) {
            const int src_idx = j * image->stride + i;
            const uint8_t alpha = image->bitmap[src_idx];

            #define PLANE_DST(plane, x, y) (plane.ptr[0][(y) * plane.pitch[0] + (x)])
            PLANE_DST(planeY, i + x_offset, j + y_offset) = subY;
            PLANE_DST(planeU, i + x_offset, j + y_offset) = subU;
            PLANE_DST(planeV, i + x_offset, j + y_offset) = subV;
            PLANE_DST(planeA, i + x_offset, j + y_offset) = (uint8_t)clamp(((int)subA * alpha) >> 8, 0, 255);
            #undef PLANE_DST
        }
    }
    //GPUへ転送
    auto frame = std::make_unique<CUFrameBuf>(bufCPU->frame.width, bufCPU->frame.height, bufCPU->frame.csp);
    err = frame->alloc();
    if (err != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("Failed to allocate device memory for subtitle image %dx%d: %s.\n"), image->w, image->h, get_err_mes(err));
        return SubImageData(
            std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
            std::unique_ptr<CUFrameBuf>(), 0, 0);
    }
    frame->copyFrameAsync(&bufCPU->frame, stream);
    return SubImageData(std::move(frame), std::unique_ptr<CUFrameBuf>(), std::move(bufCPU), image->dst_x, image->dst_y);
}

RGY_ERR NVEncFilterSubburn::procFrameText(RGYFrameInfo *pOutputFrame, int64_t frameTimeMs, hipStream_t stream) {
    int nDetectChange = 0;
    const auto frameImages = ass_render_frame(m_assRenderer.get(), m_assTrack.get(), frameTimeMs, &nDetectChange);

    if (!frameImages) {
        m_subImages.clear();
    } else if (nDetectChange) {
        m_subImages.clear();
        for (auto image = frameImages; image; image = image->next) {
            m_subImages.push_back(textRectToImage(image, stream));
        }
    }
    auto prm = std::dynamic_pointer_cast<NVEncFilterParamSubburn>(m_param);
    if (!prm) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (m_subImages.size()) {
        static const std::map<RGY_CSP, decltype(proc_frame<uint8_t, 8>) *> func_list ={
            { RGY_CSP_YV12,      proc_frame<uint8_t,   8> },
            { RGY_CSP_YV12_16,   proc_frame<uint16_t, 16> },
            { RGY_CSP_YUV444,    proc_frame<uint8_t,   8> },
            { RGY_CSP_YUV444_16, proc_frame<uint16_t, 16> }
        };
        if (func_list.count(pOutputFrame->csp) == 0) {
            AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pOutputFrame->csp]);
            return RGY_ERR_UNSUPPORTED;
        }
        for (uint32_t irect = 0; irect < m_subImages.size(); irect++) {
            const RGYFrameInfo *pSubImg = &m_subImages[irect].image->frame;
            auto sts = func_list.at(pOutputFrame->csp)(pOutputFrame, pSubImg, m_subImages[irect].x, m_subImages[irect].y,
                prm->subburn.transparency_offset, prm->subburn.brightness, prm->subburn.contrast, stream);
            if (sts != RGY_ERR_NONE) {
                AddMessage(RGY_LOG_ERROR, _T("error at subburn(%s): %s.\n"),
                    RGY_CSP_NAMES[pOutputFrame->csp],
                    get_err_mes(sts));
                return sts;
            }
        }
    }
    return RGY_ERR_NONE;
}

SubImageData NVEncFilterSubburn::bitmapRectToImage(const AVSubtitleRect *rect, const RGYFrameInfo *outputFrame, const sInputCrop &crop, hipStream_t stream) {
    //YUV420の関係で縦横2pixelずつ処理するので、2で割り切れている必要がある
    const int x_offset = ((rect->x % 2) != 0) ? 1 : 0;
    const int y_offset = ((rect->y % 2) != 0) ? 1 : 0;
    RGYFrameInfo img;
    img.csp = RGY_CSP_YUVA444;
    img.width  = ALIGN(rect->w + x_offset, 2);
    img.height = ALIGN(rect->h + y_offset, 2);
    img.mem_type = RGY_MEM_TYPE_CPU;
    img.picstruct = RGY_PICSTRUCT_FRAME;
    auto bufCPU = std::make_unique<CUFrameBuf>(img);
    auto err = bufCPU->allocHost();
    if (err != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("Failed to allocate host memory for subtitle image %dx%d: %s.\n"), rect->w, rect->h, get_err_mes(err));
        return SubImageData(
            std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
            std::unique_ptr<CUFrameBuf>(), 0, 0);
    }

    auto planeY = getPlane(&bufCPU->frame, RGY_PLANE_Y);
    auto planeU = getPlane(&bufCPU->frame, RGY_PLANE_U);
    auto planeV = getPlane(&bufCPU->frame, RGY_PLANE_V);
    auto planeA = getPlane(&bufCPU->frame, RGY_PLANE_A);

    //とりあえずすべて0で初期化しておく
    memset(planeY.ptr[0], 0, (size_t)planeY.pitch[0] * planeY.height);

    //とりあえずすべて0で初期化しておく
    //Alpha=0で透明なので都合がよい
    memset(planeA.ptr[0], 0, (size_t)planeA.pitch[0] * planeA.height);

    for (int j = 0; j < planeU.height; j++) {
        auto ptr = planeU.ptr[0] + j * planeU.pitch[0];
        for (int i = 0; i < planeU.pitch[0] / (int)sizeof(ptr[0]); i++) {
            ptr[i] = 128;
        }
    }
    for (int j = 0; j < planeV.height; j++) {
        auto ptr = planeV.ptr[0] + j * planeV.pitch[0];
        for (int i = 0; i < planeV.pitch[0] / (int)sizeof(ptr[0]); i++) {
            ptr[i] = 128;
        }
    }

    //色テーブルをRGBA->YUVAに変換
    const uint32_t *pColorARGB = (uint32_t *)rect->data[1];
    alignas(32) uint32_t colorTableYUVA[256];
    memset(colorTableYUVA, 0, sizeof(colorTableYUVA));

    const uint32_t nColorTableSize = rect->nb_colors;
    assert(nColorTableSize <= _countof(colorTableYUVA));
    for (uint32_t ic = 0; ic < nColorTableSize; ic++) {
        const uint32_t subColor = pColorARGB[ic];
        const uint8_t subA = (uint8_t)(subColor >> 24);
        const uint8_t subR = (uint8_t)((subColor >> 16) & 0xff);
        const uint8_t subG = (uint8_t)((subColor >>  8) & 0xff);
        const uint8_t subB = (uint8_t)(subColor        & 0xff);

        const uint8_t subY = (uint8_t)clamp((( 66 * subR + 129 * subG +  25 * subB + 128) >> 8) +  16, 0, 255);
        const uint8_t subU = (uint8_t)clamp(((-38 * subR -  74 * subG + 112 * subB + 128) >> 8) + 128, 0, 255);
        const uint8_t subV = (uint8_t)clamp(((112 * subR -  94 * subG -  18 * subB + 128) >> 8) + 128, 0, 255);

        colorTableYUVA[ic] = ((subA << 24) | (subV << 16) | (subU << 8) | subY);
    }

    //YUVで字幕の画像データを構築
    for (int j = 0; j < rect->h; j++) {
        for (int i = 0; i < rect->w; i++) {
            const int src_idx = j * rect->linesize[0] + i;
            const int ic = rect->data[0][src_idx];

            const uint32_t subColor = colorTableYUVA[ic];
            const uint8_t subA = (uint8_t)(subColor >> 24);
            const uint8_t subV = (uint8_t)((subColor >> 16) & 0xff);
            const uint8_t subU = (uint8_t)((subColor >>  8) & 0xff);
            const uint8_t subY = (uint8_t)(subColor        & 0xff);

            #define PLANE_DST(plane, x, y) (plane.ptr[0][(y) * plane.pitch[0] + (x)])
            PLANE_DST(planeY, i + x_offset, j + y_offset) = subY;
            PLANE_DST(planeU, i + x_offset, j + y_offset) = subU;
            PLANE_DST(planeV, i + x_offset, j + y_offset) = subV;
            PLANE_DST(planeA, i + x_offset, j + y_offset) = subA;
            #undef PLANE_DST
        }
    }

    //GPUへ転送
    auto frameTemp = std::make_unique<CUFrameBuf>(bufCPU->frame.width, bufCPU->frame.height, bufCPU->frame.csp);
    err = frameTemp->alloc();
    if (err != RGY_ERR_NONE) {
        AddMessage(RGY_LOG_ERROR, _T("Failed to allocate device memory for subtitle image %dx%d: %s.\n"), rect->w, rect->h, get_err_mes(err));
        return SubImageData(
            std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
            std::unique_ptr<CUFrameBuf>(), 0, 0);
    }
    frameTemp->copyFrameAsync(&bufCPU->frame, stream);
    auto prm = std::dynamic_pointer_cast<NVEncFilterParamSubburn>(m_param);

    decltype(frameTemp) frame;
    if (prm->subburn.scale == 1.0f) {
        frame = std::move(frameTemp);
    } else {
#if 0
        RGYFrameInfo tempframe = img;
        std::vector<uint8_t> temp(imgInfoEx.frame_size);
        memcpy(temp.data(), img.ptr, temp.size());
        tempframe.ptr = temp.data();
        auto tmpY = getPlane(&tempframe, RGY_PLANE_Y);
        auto tmpU = getPlane(&tempframe, RGY_PLANE_U);
        auto tmpV = getPlane(&tempframe, RGY_PLANE_V);
        for (int j = 0; j < rect->h; j++) {
            for (int i = 0; i < rect->w; i++) {
                #define IDX(x, y) ((clamp(y,0,rect->h)+y_offset) * img.pitch + (clamp(x,0,rect->w)+x_offset))
                const int dst_idx = IDX(i,j);
                if (planeA.ptr[dst_idx] == 0) {
                    int minidx = -1;
                    uint8_t minval = 255;
                    for (int jy = -1; jy <= 1; jy++) {
                        for (int ix = -1; ix <= 1; ix++) {
                            int idx = IDX(i+ix, j+jy);
                            if (planeA.ptr[idx] != 0) {
                                auto val = tmpY.ptr[idx];
                                if (val < minval) {
                                    minidx = idx;
                                    minval = val;
                                }
                            }
                        }
                    }
                    if (minidx >= 0) {
                        planeY.ptr[dst_idx] = tmpY.ptr[minidx];
                        planeU.ptr[dst_idx] = tmpU.ptr[minidx];
                        planeV.ptr[dst_idx] = tmpV.ptr[minidx];
                    }
                }
                #undef IDX
            }
        }
#endif

        frame = std::make_unique<CUFrameBuf>(
            ALIGN((int)(bufCPU->frame.width  * prm->subburn.scale + 0.5f), 4),
            ALIGN((int)(bufCPU->frame.height * prm->subburn.scale + 0.5f), 4), bufCPU->frame.csp);
        err = frame->alloc();
        if (err != RGY_ERR_NONE) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to allocate device memory for scaled subtitle image %dx%d: %s.\n"), frame->width(), frame->height(), get_err_mes(err));
            return SubImageData(
                std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
                std::unique_ptr<CUFrameBuf>(), 0, 0);
        }
        unique_ptr<NVEncFilterResize> filterResize(new NVEncFilterResize());
        shared_ptr<NVEncFilterParamResize> paramResize(new NVEncFilterParamResize());
        paramResize->frameIn = frameTemp->frame;
        paramResize->frameOut = frame->frame;
        paramResize->baseFps = prm->baseFps;
        paramResize->frameOut.mem_type = RGY_MEM_TYPE_GPU;
        paramResize->bOutOverwrite = false;
        paramResize->interp = RGY_VPP_RESIZE_BILINEAR;
        filterResize->init(paramResize, m_pLog);
        m_resize = std::move(filterResize);

        int filterOutputNum = 0;
        RGYFrameInfo *filterOutput[1] = { &frame->frame };
        m_resize->filter(&frameTemp->frame, (RGYFrameInfo **)&filterOutput, &filterOutputNum, stream);
    }
    int x_pos = ALIGN((int)(prm->subburn.scale * rect->x + 0.5f) - ((crop.e.left + crop.e.right) / 2), 2);
    int y_pos = ALIGN((int)(prm->subburn.scale * rect->y + 0.5f) - crop.e.up - crop.e.bottom, 2);
    if (m_outCodecDecodeCtx->height > 0) {
        const double y_factor = rect->y / (double)m_outCodecDecodeCtx->height;
        y_pos = ALIGN((int)(outputFrame->height * y_factor + 0.5f), 2);
        y_pos = std::min(y_pos, outputFrame->height - rect->h);
    }
    return SubImageData(std::move(frame), std::move(frameTemp), std::move(bufCPU), x_pos, y_pos);
}


RGY_ERR NVEncFilterSubburn::procFrameBitmap(RGYFrameInfo *pOutputFrame, const int64_t frameTimeMs, const sInputCrop &crop, const bool forced_subs_only, hipStream_t stream) {
    if (m_subData) {
        if (m_subData->num_rects != m_subImages.size()) {
            for (uint32_t irect = 0; irect < m_subData->num_rects; irect++) {
                const AVSubtitleRect *rect = m_subData->rects[irect];
                if (forced_subs_only && !(rect->flags & AV_SUBTITLE_FLAG_FORCED)) {
                    AddMessage(RGY_LOG_DEBUG, _T("skipping non-forced sub at %s\n"), getTimestampString(frameTimeMs, av_make_q(1, 1000)).c_str());
                    // 空の値をいれる
                    m_subImages.push_back(SubImageData(
                        std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
                        std::unique_ptr<CUFrameBuf>(), 0, 0));
                } else if (rect->w == 0 || rect->h == 0) {
                    // 空の値をいれる
                    m_subImages.push_back(SubImageData(
                        std::unique_ptr<CUFrameBuf>(), std::unique_ptr<CUFrameBuf>(),
                        std::unique_ptr<CUFrameBuf>(), 0, 0));
                } else {
                    m_subImages.push_back(bitmapRectToImage(rect, pOutputFrame, crop, stream));
                }
            }
        }
        if ((m_subData->num_rects != m_subImages.size())) {
            AddMessage(RGY_LOG_ERROR, _T("unexpected error.\n"));
            return RGY_ERR_UNKNOWN;
        }
        auto prm = std::dynamic_pointer_cast<NVEncFilterParamSubburn>(m_param);
        if (!prm) {
            AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
            return RGY_ERR_INVALID_PARAM;
        }
        static const std::map<RGY_CSP, decltype(proc_frame<uint8_t, 8>) *> func_list = {
            { RGY_CSP_YV12,      proc_frame<uint8_t,   8> },
            { RGY_CSP_YV12_16,   proc_frame<uint16_t, 16> },
            { RGY_CSP_YUV444,    proc_frame<uint8_t,   8> },
            { RGY_CSP_YUV444_16, proc_frame<uint16_t, 16> }
        };
        if (func_list.count(pOutputFrame->csp) == 0) {
            AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pOutputFrame->csp]);
            return RGY_ERR_UNSUPPORTED;
        }
        for (uint32_t irect = 0; irect < m_subImages.size(); irect++) {
            if (m_subImages[irect].image) {
                const RGYFrameInfo *pSubImg = &m_subImages[irect].image->frame;
                auto sts = func_list.at(pOutputFrame->csp)(pOutputFrame, pSubImg, m_subImages[irect].x, m_subImages[irect].y,
                    prm->subburn.transparency_offset, prm->subburn.brightness, prm->subburn.contrast, stream);
                if (sts != RGY_ERR_NONE) {
                    AddMessage(RGY_LOG_ERROR, _T("error at subburn(%s): %s.\n"),
                        RGY_CSP_NAMES[pOutputFrame->csp],
                        get_err_mes(sts));
                    return sts;
                }
            }
        }
    }
    return RGY_ERR_NONE;
}

#endif //#if ENABLE_AVSW_READER
